#include "hip/hip_runtime.h"
#include <algorithm>//All code done by Oleg K
#include <iostream>
#include <fstream>
#include <sstream>
#include <utility>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <string>
#include <cmath>
#include <map>
#include <ctime>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include "hip/hip_runtime.h"
#include ""
//#define WIN32_LEAN_AND_MEAN
#include <Windows.h>
#include <MMSystem.h>
#pragma comment(lib, "winmm.lib")
#define _CRTDBG_MAP_ALLOC
#include <crtdbg.h>
using namespace std;

#define _DTH hipMemcpyDeviceToHost
#define _DTD hipMemcpyDeviceToDevice
#define _HTD hipMemcpyHostToDevice
#define INF (1<<28)
#define THREADS 256
#define SMALL_THREADS 64
#define DO_GPU 1

#define NUM_BOXES 50
#define NUM_COLORS 20

//Unthinking respect for authority is the greatest enemy of truth

bool InitMMTimer(UINT wTimerRes);
void DestroyMMTimer(UINT wTimerRes, bool init);

void _gen_random(int *AM, const int num_boxes,const int num_colors);

int cpu_version(const int *AM,const int num_boxes, const int num_colors,const int p_space){
	int ret=INF;
	int *Cost=(int *)malloc(num_boxes*num_colors*sizeof(int));
	int *DP=(int *)malloc(p_space*sizeof(int));
	memset(Cost,0,num_boxes*num_colors*sizeof(int));
	for(int i=0;i<num_boxes;i++)for(int j=0;j<num_colors;j++)for(int ii=0;ii<num_boxes;ii++)if(i!=ii){
		Cost[i*num_colors+j]+=AM[ii*num_colors+j];
	}
	for(int i=0;i<=num_boxes;i++)for(int j=0;j<(1<<num_colors);j++){
		DP[i*(1<<num_colors)+j]= (i==num_boxes && j==((1<<num_colors)-1)) ? 0:INF;
	}
	int idx;
	for(int i=num_boxes-1;i>=0;i--){
		for(int j=0;j<(1<<num_colors);j++){
			idx=i*(1<<num_colors)+j;
			DP[idx]=min(DP[idx],DP[(i+1)*(1<<num_colors)+j]);
			for(int k=0;k<num_colors;k++)if(!(j&(1<<k))){
				DP[idx]=min(DP[idx],Cost[i*num_colors+k]+DP[((i+1)*(1<<num_colors))+(j|(1<<k))]);
			}
		}
	}
	ret=DP[0];
	free(Cost);
	free(DP);
	return ret;
}

__global__ void mem_setup(int *DP, const int num_boxes, const int num_colors){//This can be sped up, PM me for details
	const int i=blockIdx.y;
	const int j=threadIdx.x+blockIdx.x*blockDim.x;
	const int m_bound=(1<<num_colors);
	if(j<m_bound){
		DP[i*m_bound+j]= (i==num_boxes && j==(m_bound-1)) ? 0:INF;
	}
}
__global__ void Cost_setup(const int *AM, int *Cost,const int num_boxes, const int num_colors){
	const int i=threadIdx.x+blockIdx.x*blockDim.x;
	const int j=blockIdx.y;
	const int k=blockIdx.z;
	if(i<num_boxes && i!=k){
		atomicAdd(&Cost[i*num_colors+j],AM[k*num_colors+j]);
	}
}
__global__ void GPU_version(const int *Cost, int *DP, const int ii, const int num_boxes, const int num_colors){
	const int j=threadIdx.x+blockIdx.x*blockDim.x;
	const int k=blockIdx.y;
	const int m_bound=(1<<num_colors);

	__shared__ int cur_cost;
	if(threadIdx.x==0){//Singapore person, I see you and think since you look at my code so often you should follow or star my work
		cur_cost=Cost[ii*num_colors+k];
	}
	__syncthreads();

	if(j<m_bound){
		const int idx=ii*m_bound+j;
		atomicMin(&DP[idx],DP[(ii+1)*m_bound+j]);
		if(!(j&(1<<k))){
			atomicMin(&DP[idx],cur_cost+DP[(ii+1)*m_bound+(j|(1<<k))]);
		}
	}
}

int main(){
	char ch;
	srand(time(NULL));
	const int num_boxes=NUM_BOXES;
	const int num_colors=NUM_COLORS;
	
	const int problem_space=(num_boxes+1)*(1<<num_colors);
	
	int *AM=(int *)malloc(num_boxes*num_colors*sizeof(int));

	_gen_random(AM,num_boxes,num_colors);

	int CPU_ans=INF,GPU_ans=INF;

	cout<<"\nRunning CPU implementation..\n";
    UINT wTimerRes = 0;
	DWORD CPU_time=0,GPU_time=0;
    bool init = InitMMTimer(wTimerRes);
    DWORD startTime=timeGetTime();

	CPU_ans=cpu_version(AM,num_boxes,num_colors,problem_space);

	DWORD endTime = timeGetTime();
    CPU_time=endTime-startTime;
    cout<<"CPU solution timing: "<<CPU_time<< " , answer= "<<CPU_ans<<'\n';
    DestroyMMTimer(wTimerRes, init);

	int compute_capability=0;
    hipDeviceProp_t deviceProp;
    hipError_t err=hipGetDeviceProperties(&deviceProp, compute_capability);
    if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
    string ss= (deviceProp.major>=3 && deviceProp.minor>=5) ? "Capable!\n":"Not Sufficient compute capability!\n";
    cout<<ss;

	if(DO_GPU && (deviceProp.major>=3 && deviceProp.minor>=5)){// !(AMD || PYTHON || RUBY)
		const int num_bytes=problem_space*sizeof(int);
		const int num_bytesAM=num_boxes*num_colors*sizeof(int);
		const int m_bound=(1<<num_colors);
		int *D_AM,*D_Cost,*D_DP;
		err=hipMalloc((void**)&D_AM,num_bytesAM);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipMalloc((void**)&D_Cost,num_bytesAM);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipMalloc((void**)&D_DP,num_bytes);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		dim3 memGrid((m_bound+THREADS-1)/THREADS,(num_boxes+1),1);
		dim3 costGrid((num_boxes+SMALL_THREADS-1)/SMALL_THREADS,num_colors,num_boxes);
		dim3 dpGrid((m_bound+THREADS-1)/THREADS,num_colors,1);
		int ii=num_boxes-1;
		//there is one thing we do know: that man is here for the sake of other men
		wTimerRes = 0;
        init = InitMMTimer(wTimerRes);
        startTime = timeGetTime();

		err=hipMemset(D_Cost,0,num_bytesAM);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		
		err=hipMemcpy(D_AM,AM,num_bytesAM,_HTD);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		mem_setup<<<memGrid,THREADS>>>(D_DP,num_boxes,num_colors);
		err = hipDeviceSynchronize();
        if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		Cost_setup<<<costGrid,SMALL_THREADS>>>(D_AM,D_Cost,num_boxes,num_colors);
		err = hipDeviceSynchronize();
        if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		for(;ii>=0;ii--){
			GPU_version<<<dpGrid,THREADS>>>(D_Cost,D_DP,ii,num_boxes,num_colors);
			err = hipDeviceSynchronize();
			if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		}
		err=hipMemcpy(&GPU_ans,D_DP,sizeof(int),_DTH);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

		endTime = timeGetTime();
        GPU_time=endTime-startTime;
        cout<<"CUDA timing: "<<GPU_time<<" , answer= "<<GPU_ans<<'\n';
        DestroyMMTimer(wTimerRes, init);

		err=hipFree(D_AM);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipFree(D_Cost);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
		err=hipFree(D_DP);
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	}

	free(AM);

	cin>>ch;
	return 0;
}

bool InitMMTimer(UINT wTimerRes){
	TIMECAPS tc;
	if (timeGetDevCaps(&tc, sizeof(TIMECAPS)) != TIMERR_NOERROR) {return false;}
	wTimerRes = min(max(tc.wPeriodMin, 1), tc.wPeriodMax);
	timeBeginPeriod(wTimerRes); 
	return true;
}

void DestroyMMTimer(UINT wTimerRes, bool init){
	if(init)
		timeEndPeriod(wTimerRes);
}

void _gen_random(int *AM, const int num_boxes,const int num_colors){
	for(int i=0;i<num_boxes;i++){
		for(int j=0;j<num_colors;j++){
			AM[i*num_colors+j]= (rand()%4==0) ? 0:((rand()%127)+1);
		}
	}
}









